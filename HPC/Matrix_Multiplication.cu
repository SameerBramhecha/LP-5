
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;


// CUDA code to multiply matrices
__global__ void multiply_gpu(int* A, int* B, int* C, int size) {
    // Uses thread indices and block indices to compute each element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}

__global__ void matrix_multiplication_cpu(int *a, int *b, int *c, int size){
    for(int i = 0; i <size; i++){
        for(int j = 0; j < size; j++){
            int sum = 0;
            for(int k = 0; k < size; k++){
                sum += a[i*size + k] * b[k*size + j];
            }
            c[i*size + j] = sum;
        }
    }
}

void initialize(int* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = rand() % 10;
    }
}


void print(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            cout << matrix[row * size + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}


int main() {
    int* A, * B, * C;

    int N = 3;
    int blockSize =  16;

    int matrixSize = N * N;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = new int[matrixSize];
    B = new int[matrixSize];
    C = new int[matrixSize];

    initialize(A, N);
    initialize(B, N);
    cout << "Matrix A: \n";
    print(A, N);

    cout << "Matrix B: \n";
    print(B, N);

    
    int* X, * Y, * Z;
    // Allocate space
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, matrixBytes);
    hipMalloc(&Z, matrixBytes);

    // Copy values from A to X
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);
    
    // Copy values from A to X and B to Y
    hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 2;

    // Blocks per grid dimension
    int BLOCKS = N + THREADS - 1 / THREADS;

    hipEvent_t start,stop;
    float elapsedTime;

    // Use dim3 structs for block  and grid dimensions
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

     hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // Launch kernel
    multiply_gpu<<<blocks, threads>>>(X, Y, Z, N);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    cout << "Multiplication of matrix A and B: \n";
    print(C, N);
cout<<"Elapsed Time : "<<elapsedTime<<endl;
    

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // Launch kernel
    matrix_multiplication_cpu<<<blocks, threads>>>(X, Y, Z, N);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    cout << "Multiplication of matrix A and B: \n";
    print(C, N);
cout<<"Elapsed Time : "<<elapsedTime<<endl;
    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}
