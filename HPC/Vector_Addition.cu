/*
This code contains comparison using time for CPU And GPU.
*/


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

void initialize(int *vector,int size){
    for(int i=0;i<size;i++){
        vector[i] = rand()%10;
    }
}

void print(int *vector,int size){
  for(int i=0;i<size;i++){
      cout<<vector[i] << " ";
  }
  cout<<endl;
}

__global__ void add_cpu(int *A, int *B,int *C,int size){
    for(int i=0;i<size;i++){
        C[i] = A[i] + B[i];
    }
}

__global__ void add_gpu(int *A, int *B,int *C,int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < size){
        C[tid] = A[tid] + B[tid];
    }
}

int main(){
    int N = 5;
    int vectorSize = N;
    size_t vectorBytes= vectorSize * sizeof(int);

    int *A,*B,*C;

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A,N);
    initialize(B,N);

    cout<<"Vector A : ";
    print(A,N);

    cout<<"Vector B : ";
    print(B,N);

    int *X,*Y,*Z;
    hipMalloc(&X,vectorBytes);
    hipMalloc(&Y,vectorBytes);
    hipMalloc(&Z,vectorBytes);

    hipMemcpy(X,A,vectorBytes,hipMemcpyHostToDevice);
    hipMemcpy(Y,B,vectorBytes,hipMemcpyHostToDevice);
    
    hipEvent_t start,stop;
    float elapsedTime;

    int threadsPerBlock = 256;
    int blocksPerGrid = N + threadsPerBlock -1 / threadsPerBlock;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    add_gpu<<<blocksPerGrid, threadsPerBlock>>>(X,Y,Z,N);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(C,Z,vectorBytes,hipMemcpyDeviceToHost);
    cout<<"GPU RESULT : ";
    print(C,N); 
    cout<<"Elapsed Time : "<<elapsedTime<<endl;

    //cpu
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    add_cpu<<<blocksPerGrid, threadsPerBlock>>>(X,Y,Z,N);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(C,Z,vectorBytes,hipMemcpyDeviceToHost);
    cout<<"CPU RESULT : ";
    print(C,N); 
    cout<<"Elapsed Time : "<<elapsedTime<<endl;

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}
